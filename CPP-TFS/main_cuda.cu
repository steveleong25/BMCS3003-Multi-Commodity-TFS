#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <utility>
#include <cmath>
#include "NetworkGraph.hpp"
#include "PathFinder.hpp"
#include <hip/hip_runtime.h>

__device__ __host__ void cudaSendFlow(NetworkGraph& graph, const std::vector<std::string>& path, double amount) {
    for (int i = 0; i < path.size() - 1; ++i) {
        Edge& edge = graph.getEdge(path[i], path[i + 1]);
        edge.flow += amount;
    }
}

__global__ void calculateSuccessRates(double* successRates, double* unitsDelivered, double* demands, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        successRates[idx] = (demands[idx] > 0) ? unitsDelivered[idx] / demands[idx] : 1.0;
    }
}

__global__ void findLowestSuccessRate(double* successRates, int* lowestIndex, int size) {
    __shared__ int localIndex;
    __shared__ double localMin;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadIdx.x == 0) {
        localIndex = 0;
        localMin = successRates[0];
    }

    __syncthreads();

    if (idx < size) {
        if (successRates[idx] < localMin) {
            atomicMin(&localIndex, idx);
        }
    }

    __syncthreads();

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *lowestIndex = localIndex;
    }
}

extern "C" void CUDA_equalDistributionAlgorithm(NetworkGraph & graph,
    std::vector<std::pair<std::string, std::string>> commodities,
    std::vector<double> demands) {

    int size = commodities.size();

    // Allocate host memory
    std::vector<double> successRates(size, 0.0);
    std::vector<double> unitsDelivered(size, 0.0);

    // Allocate device memory
    double* d_successRates, * d_unitsDelivered, * d_demands;
    int* d_lowestIndex;

    hipMalloc(&d_successRates, size * sizeof(double));
    hipMalloc(&d_unitsDelivered, size * sizeof(double));
    hipMalloc(&d_demands, size * sizeof(double));
    hipMalloc(&d_lowestIndex, sizeof(int));

    hipMemcpy(d_demands, demands.data(), size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_unitsDelivered, unitsDelivered.data(), size * sizeof(double), hipMemcpyHostToDevice);

    bool moreFlowNeeded = true;

    while (moreFlowNeeded) {
        moreFlowNeeded = false;

        // Calculate success rates in parallel
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

        calculateSuccessRates << <blocksPerGrid, threadsPerBlock >> > (d_successRates, d_unitsDelivered, d_demands, size);
        hipDeviceSynchronize();

        // Find commodity with the lowest success rate in parallel
        findLowestSuccessRate << <blocksPerGrid, threadsPerBlock >> > (d_successRates, d_lowestIndex, size);
        hipDeviceSynchronize();

        int lowestIndex;
        hipMemcpy(&lowestIndex, d_lowestIndex, sizeof(int), hipMemcpyDeviceToHost);

        const std::string& source = commodities[lowestIndex].first;
        const std::string& destination = commodities[lowestIndex].second;
        double remainingDemand = demands[lowestIndex] - unitsDelivered[lowestIndex];

        // Get all paths from source to destination (host-side operation for now)
        std::vector<std::vector<std::string>> allPaths = findAllPaths(graph.getEdges(), source, destination);

        for (const auto& path : allPaths) {
            if (remainingDemand <= 0) break;

            // Calculate bottleneck capacity
            double pathCapacity = std::numeric_limits<double>::max();
            for (size_t j = 0; j < path.size() - 1; ++j) {
                Edge& edge = graph.getEdge(path[j], path[j + 1]);
                pathCapacity = std::min(pathCapacity, static_cast<double>(edge.capacity - edge.flow));
            }

            if (pathCapacity > 0) {
                moreFlowNeeded = true;

                // Allocate flow
                double flowToSend = std::min(remainingDemand, pathCapacity);
                cudaSendFlow(graph, path, flowToSend);

                // Update demand and track delivered flow
                unitsDelivered[lowestIndex] += flowToSend;
                remainingDemand -= flowToSend;
            }
        }

        hipMemcpy(d_unitsDelivered, unitsDelivered.data(), size * sizeof(double), hipMemcpyDeviceToHost);
    }

    // Free device memory
    hipFree(d_successRates);
    hipFree(d_unitsDelivered);
    hipFree(d_demands);
    hipFree(d_lowestIndex);

    std::cout << "CUDA-based proportional flow algorithm completed.\n";
}
